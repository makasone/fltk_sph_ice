#include "hip/hip_runtime.h"
//�t�̂ƌő̂̉^���v�Z���ʂ��Ԃ��C�ŏI�I�Ȍ��ʂ��v�Z
//���͒P���ɐ��`���

#ifndef _GPU_ICE_INTERPOLATION_H_
#define _GPU_ICE_INTERPOLATION_H_

#include <iostream>

using namespace std;

#define EDGE 17
//#define EDGE 27

void LaunchInterPolationGPU(int prtNum, float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel);

__global__ void LinerInterPolation(float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel);

void LaunchInterPolationGPU(int prtNum, float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel)
{	//cout << __FUNCTION__ << endl;

	int n = pow(prtNum, 1.0/3.0) + 0.5;	//�����̂̂P�ӂ̒��_��

	dim3 grid(n, n);
	dim3 block(n, 1, 1);

	//���`���
	LinerInterPolation<<<grid ,block>>>(sldPrtPos, sldPrtVel, sphPrtPos, sphPrtVel);

	hipDeviceSynchronize();
}

__global__
	void LinerInterPolation(float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel)
{
	//�v�Z���闱�q�̔���
	int pIndx = blockIdx.x * EDGE * EDGE + blockIdx.y * EDGE + threadIdx.x;

	//���`���
	sphPrtPos[pIndx*4+0] = sldPrtPos[pIndx*3+0];
	sphPrtPos[pIndx*4+1] = sldPrtPos[pIndx*3+1];
	sphPrtPos[pIndx*4+2] = sldPrtPos[pIndx*3+2];

	sphPrtVel[pIndx*4+0] = sldPrtVel[pIndx*3+0];
	sphPrtVel[pIndx*4+1] = sldPrtVel[pIndx*3+1];
	sphPrtVel[pIndx*4+2] = sldPrtVel[pIndx*3+2];
}





#endif