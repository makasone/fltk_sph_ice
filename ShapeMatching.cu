#include "hip/hip_runtime.h"
#ifndef _GPU_SHAPE_MATCHING_H_
#define _GPU_SHAPE_MATCHING_H_

#include <math.h>	//�g����H
#include <stdio.h>

//#include <cstdio>
//#include <cmath>
//#include <cstdlib>
//
//#include "rx_utility.h"
//#include <rx_matrix.h>
//
//#include "rx_nnsearch.h"

//#include <rx_cu_common.cu>
#include <rx_cu_common.cuh>	//�搶����`�����֗��@�\���g����@�Ȃ����A�C���N���[�h����ƐԂ��Ȃ�Ȃ�

#define SM_DIM 3

//�����n���ϐ������Ԉ���Ă��Ă��G���[���o�Ȃ��̂Œ���

void LaunchShapeMathcingGPU(float* prtPos, float* prtVel, float* orgPos, float* curPos, float* vel, int* pIndxes, float dt, int prtNum);
__global__ void Update(float* prtPos, float* prtVel, float* orgPos, float* curPos, float* vel, int* pIndxes, float dt, int prtNum);
__device__ void ExternalForce(float* prtPos, float* prtVel, float* curPos, float* vel, int* pIndxes, float dt, int prtNum);
__device__ void ProjectPos(float* prtPos, float* prtVel, float* orgPos, float* curPos, float* vel, int* pIndxes, float dt, int prtNum);
__device__ void Integrate(float* prtPos, float* prtVel, float* curPos, float* vel, int* pIndxes, float dt, int prtNum);

__device__ void PolarDecomposition(matrix3x3 &A, matrix3x3 &R, matrix3x3 &S);

//�s�񉉎Z
__device__ void MakeIdentity(matrix3x3 &M);
__device__ matrix3x3 Transpose(const matrix3x3 &M);
__device__ matrix3x3 Inverse(const matrix3x3 &M);

__device__ matrix3x3 Multiple(matrix3x3 &M1, matrix3x3 &M2);
__device__ float3 Multiple(matrix3x3 &M1, float3& V);

//GPU����
void LaunchShapeMatchingGPU(float* prtPos, float* prtVel, float* orgPos, float* curPos, float* vel, int* pIndxes, float dt, int prtNum)
{
	
	//printf("LaunchGPUKernel");

	dim3 grid(1, 1);
	dim3 block(1, 1, 1);

	//�^���v�Z
	Update <<< grid , block >>> (prtPos, prtVel, orgPos, curPos, vel, pIndxes, dt, prtNum);
}


//GPU�̈ʒu�E���x�X�V
__global__
void Update(
	float* prtPos,
	float* prtVel, 
	float* orgPos,
	float* curPos, 
	float* vel,
	int* pIndxes,
	float dt,
	int prtNum)
{
	//printf("d_Integrate\n");	//�߂��Ⴍ����o��̂Œ���

	ExternalForce(prtPos, prtVel, curPos, vel, pIndxes, dt, prtNum);
	ProjectPos(prtPos, prtVel, orgPos, curPos, vel, pIndxes, dt, prtNum);
	Integrate(prtPos, prtVel, curPos, vel, pIndxes, dt, prtNum);
}

__device__
	void ExternalForce(float* prtPos, float* prtVel, float* curPos, float* vel, int* pIndxes, float dt, int prtNum)
{
	// �d�͂̉e����t���C���x�𔽉f
	for(int i = 0; i < prtNum; ++i)
	{
		int pIndx = pIndxes[i]*4;
		int cIndx = i*SM_DIM;

		for(int j = 0; j < SM_DIM; j++)
		{
			int jpIndx = pIndx+j;
			int jcIndx = cIndx+j;

			curPos[jcIndx] = prtPos[jpIndx]+prtVel[jpIndx]*dt;
		}
	}

	// ���E�ǂ̉e��
	//���������Ȃ�d���Ȃ邪�C����͂���݂���
	//float res = 0.9;	// �����W��
	//for(int i = 0; i < prtNum; ++i){
	//	//if(m_pFix[i]) continue;
	//	//Vec3 &p = m_vCurPos[i];
	//	//Vec3 &np = m_vNewPos[i];
	//	//Vec3 &v = m_vVel[i];
	//	//if(np[0] < m_v3Min[0] || np[0] > m_v3Max[0]){
	//	//	np[0] = p[0]-v[0]*dt*res;
	//	//	np[1] = p[1];
	//	//	np[2] = p[2];
	//	//}
	//	//if(np[1] < m_v3Min[1] || np[1] > m_v3Max[1]){
	//	//	np[1] = p[1]-v[1]*dt*res;
	//	//	np[0] = p[0] ;
	//	//	np[2] = p[2];
	//	//}
	//	//if(np[2] < m_v3Min[2] || np[2] > m_v3Max[2]){
	//	//	np[2] = p[2]-v[2]*dt*res;
	//	//	np[0] = p[0];
	//	//	np[1] = p[1];
	//	//}

	//	//clamp(curPos, i*SM_DIM);
	//}
}

__device__
	void ProjectPos(float* prtPos, float* prtVel, float* orgPos, float* curPos, float* vel, int* pIndxes, float dt, int prtNum)
{
	if(prtNum <= 1) return;

	float3 cm = make_float3(0.0, 0.0, 0.0);
	float3 cm_org = make_float3(0.0, 0.0, 0.0);	// �d�S

	float mass = 0.0;	// ������

	// �d�S���W�̌v�Z
	for(int i = 0; i < prtNum;++i){
		//float m = m_pMass[i];
		float m = 1.0;
		//if(m_pFix[i]) m *= 300.0;	// �Œ�_�̎��ʂ�傫������
		
		int cIndx = i*SM_DIM;
		mass += m;

		cm.x += curPos[cIndx+0]*m;
		cm.y += curPos[cIndx+1]*m;
		cm.z += curPos[cIndx+2]*m;

		cm_org.x += orgPos[cIndx+0]*m;
		cm_org.y += orgPos[cIndx+1]*m;
		cm_org.z += orgPos[cIndx+2]*m;
	}

	cm.x /= mass;
	cm.y /= mass;
	cm.z /= mass;
	
	cm_org.x /= mass;
	cm_org.y /= mass;
	cm_org.z /= mass;

	matrix3x3 Apq, Aqq;
	float3 p, q;

	// Apq = ��mpq^T
	// Aqq = ��mqq^T
	for(int i = 0; i < prtNum; ++i)
	{
		int cIndx = i*SM_DIM;

		p.x = curPos[cIndx+0]-cm.x;
		p.y = curPos[cIndx+1]-cm.y;
		p.z = curPos[cIndx+2]-cm.z;

		q.x = orgPos[cIndx+0]-cm_org.x;
		q.y = orgPos[cIndx+1]-cm_org.y;
		q.z = orgPos[cIndx+2]-cm_org.z;

		float m = 1.0f;

		Apq.e[0].x += m*p.x*q.x;
		Apq.e[0].y += m*p.x*q.y;
		Apq.e[0].z += m*p.x*q.z;
		Apq.e[1].x += m*p.y*q.x;
		Apq.e[1].y += m*p.y*q.y;
		Apq.e[1].z += m*p.y*q.z;
		Apq.e[2].x += m*p.z*q.x;
		Apq.e[2].y += m*p.z*q.y;
		Apq.e[2].z += m*p.z*q.z;

		Aqq.e[0].x += m*q.x*q.x;
		Aqq.e[0].y += m*q.x*q.y;
		Aqq.e[0].z += m*q.x*q.z;
		Aqq.e[1].x += m*q.y*q.x;
		Aqq.e[1].y += m*q.y*q.y;
		Aqq.e[1].z += m*q.y*q.z;
		Aqq.e[2].x += m*q.z*q.x;
		Aqq.e[2].y += m*q.z*q.y;
		Aqq.e[2].z += m*q.z*q.z;
	}

	////Apq�̍s�񎮂����߁C���]���邩�𔻒�
	////�s����ȏꍇ�������̂Ł~
	////if( Apq.Determinant() < 0.0 && m_iNumVertices >= 4)
	////{
	//	//cout << "before det < 0" << endl;
	//	//�P�@�����𔽓]
	//	//Apq(0,2) = -Apq(0,2);
	//	//Apq(1,2) = -Apq(1,2);
	//	//Apq(2,2) = -Apq(2,2);

	//	//�Q�@a2��a3������
	//	//float tmp;
	//	//tmp = Apq(0,2);
	//	//Apq(0,2) = Apq(0,1);
	//	//Apq(0,1) = tmp;

	//	//tmp = Apq(1,2);
	//	//Apq(1,2) = Apq(1,1);
	//	//Apq(1,1) = tmp;

	//	//tmp = Apq(2,2);
	//	//Apq(2,2) = Apq(2,1);
	//	//Apq(2,1) = tmp;
	////}
	//

	matrix3x3 R, S;
	////PolarDecomposition(Apq, R, S, m_mtrxBeforeU);
	PolarDecomposition(Apq, R, S);

	//if(m_bLinearDeformation)
	{
		// Linear Deformations
		matrix3x3 A;
		A = Multiple(Apq, Inverse(Aqq));	// A = Apq*Aqq^-1

		//// �̐ϕۑ��̂��߂Ɂ�(det(A))�Ŋ���
		//if(m_bVolumeConservation){
		//	float det = fabs(A.Determinant());
		//	if(det > RX_FEQ_EPS){
		//		det = 1.0/sqrt(det);
		//		if(det > 2.0) det = 2.0;
		//		A *= det;
		//	}
		//}

		// �ڕW���W���v�Z���C���݂̒��_���W���ړ�
		for(int i = 0; i < prtNum; ++i){
			//if(m_pFix[i]) continue;

			int cIndx = i*SM_DIM;

			// ��]�s��R�̑���̍s��RL=��A+(1-��)R���v�Z
			q.x = orgPos[cIndx+0]-cm_org.x;
			q.y = orgPos[cIndx+1]-cm_org.y;
			q.z = orgPos[cIndx+2]-cm_org.z;

			//Vec3 gp(R*q+cm);
			float3 Rq = Multiple(R, q);
			float3 gp;
			gp.x = Rq.x + cm.x;
			gp.y = Rq.y + cm.y;
			gp.z = Rq.z + cm.z;
			
			curPos[cIndx+0] += (gp.x-curPos[cIndx+0])*1.0f/*m_dAlphas[i]*/;
			curPos[cIndx+1] += (gp.y-curPos[cIndx+1])*1.0f/*m_dAlphas[i]*/;
			curPos[cIndx+2] += (gp.z-curPos[cIndx+2])*1.0f/*m_dAlphas[i]*/;
		}
	}
}

/*!
 * ���x�ƈʒu�̍X�V
 *  - �V�����ʒu�ƌ��݂̈ʒu���W���瑬�x���Z�o
 * @param[in] dt �^�C���X�e�b�v��
 */
__device__
	void Integrate(float* prtPos, float* prtVel, float* curPos, float* vel, int* pIndxes, float dt, int prtNum)
{
	float dt1 = 1.0/dt;

	for(int i = 0; i < prtNum; ++i)
	{
		int pIndx = pIndxes[i]*4;

		for(int j = 0; j < SM_DIM; j++)
		{
			int cIndx = i*SM_DIM+j;

			vel[cIndx] = (curPos[cIndx] - prtPos[pIndx+j]) * dt1;/*+ m_v3Gravity * dt * 1.0*/;
		}
	}
	
}

__device__  
	void clamp(float* pos, int cIndx)
{
	//if(pos[cIndx+0] < m_v3Min[0]) pos[cIndx+0] = m_v3Min[0];
	//if(pos[cIndx+0] > m_v3Max[0]) pos[cIndx+0] = m_v3Max[0];
	//if(pos[cIndx+1] < m_v3Min[1]) pos[cIndx+1] = m_v3Min[1];
	//if(pos[cIndx+1] > m_v3Max[1]) pos[cIndx+1] = m_v3Max[1];
	//if(pos[cIndx+2] < m_v3Min[2]) pos[cIndx+2] = m_v3Min[2];
	//if(pos[cIndx+2] > m_v3Max[2]) pos[cIndx+2] = m_v3Max[2];
}

/*!
 * Jacobi�@�ɂ��ŗL�l�̎Z�o
 * @param[inout] a ���Ώ̍s��D�v�Z��C�Ίp�v�f�ɌŗL�l������
 * @param[out] v �ŗL�x�N�g��(a�Ɠ����T�C�Y)
 * @param[in] n �s��̃T�C�Y(n�~n)
 * @param[in] eps �����덷
 * @param[in] iter_max �ő唽����
 * @return ������
 */
__device__ 
 int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
	float *bim, *bjm;
	float bii, bij, bjj, bji;
 
	bim = new float[n];
	bjm = new float[n];
 
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < n; ++j){
			v[i*n+j] = (i == j) ? 1.0 : 0.0;
		}
	}
 
	int cnt = 0;
	for(;;){
		int i = -1, j = -1;
 
		float x = 0.0;
		for(int ia = 0; ia < n; ++ia){
			for(int ja = 0; ja < n; ++ja){
				int idx = ia*n+ja;
				if(ia != ja && fabs(a[idx]) > x){
					i = ia;
					j = ja;
					x = fabs(a[idx]);
				}
			}
		}

		if(i == -1 || j == -1) return 0;
 
		float aii = a[i*n+i];
		float ajj = a[j*n+j];
		float aij = a[i*n+j];
 
		float m_dAlpha, m_dBeta;
		m_dAlpha = (aii-ajj)/2.0;
		m_dBeta  = sqrt(m_dAlpha*m_dAlpha+aij*aij);
 
		float st, ct;
		ct = sqrt((1.0+fabs(m_dAlpha)/m_dBeta)/2.0);    // sin��
		st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*m_dBeta*ct);    // cos��
 
		// A = PAP�̌v�Z
		for(int m = 0; m < n; ++m){
			if(m == i || m == j) continue;
 
			float aim = a[i*n+m];
			float ajm = a[j*n+m];
 
			bim[m] =  aim*ct+ajm*st;
			bjm[m] = -aim*st+ajm*ct;
		}
 
		bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
		bij = 0.0;
 
		bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
		bji = 0.0;
 
		for(int m = 0; m < n; ++m){
			a[i*n+m] = a[m*n+i] = bim[m];
			a[j*n+m] = a[m*n+j] = bjm[m];
		}
		a[i*n+i] = bii;
		a[i*n+j] = bij;
		a[j*n+j] = bjj;
		a[j*n+i] = bji;
 
		// V = PV�̌v�Z
		for(int m = 0; m < n; ++m){
			float vmi = v[m*n+i];
			float vmj = v[m*n+j];
 
			bim[m] =  vmi*ct+vmj*st;
			bjm[m] = -vmi*st+vmj*ct;
		}
		for(int m = 0; m < n; ++m){
			v[m*n+i] = bim[m];
			v[m*n+j] = bjm[m];
		}
 
		float e = 0.0;
		for(int ja = 0; ja < n; ++ja){
			for(int ia = 0; ia < n; ++ia){
				if(ia != ja){
					e += fabs(a[ja*n+ia]);
				}
			}
		}
		if(e < eps) break;
 
		cnt++;
		if(cnt > iter_max) break;
	}
 
	delete [] bim;
	delete [] bjm;
 
	return cnt;
}


/*!
 * �ɕ����ŉ�]�s��ƑΏ̍s��ɕ��� A=RS
 * @param[in] A ���͍s��
 * @param[out] R ��]�s��(�����s�� R^-1 = R^T)
 * @param[out] S �Ώ̍s��
 */
__device__ 
	void PolarDecomposition(matrix3x3 &A, matrix3x3 &R, matrix3x3 &S)
{
	// S = (A^T A)^(1/2)�����߂�
	matrix3x3 ATA;
	ATA = Multiple(Transpose(A), A);	// (A^T A)�̌v�Z

	MakeIdentity(R);

	// (A^T A)���ŗL�l�������đΊp�s��ƒ����s������߂�
	// M^(1/2) = U^T M' U 
	//  M = (A^T A), M':�Ίp�s��̕����������������, U:�����s��

	//�s���float�ɕϊ�
	float* pATA = new float[9];
	float* pU = new float[9];

	for(int i = 0; i < 3; i++)
	{
		int indx = i*3;
		pATA[indx+0] = ATA.e[i].x;
		pATA[indx+1] = ATA.e[i].y;
		pATA[indx+2] = ATA.e[i].z;
	}

	EigenJacobiMethod(pATA, pU, 3);

	//float*���s��ɕϊ�	
	matrix3x3 U;

	for(int i = 0; i < 3; i++)
	{
		int indx = i*3;
		ATA.e[i].x = pATA[indx+0];
		ATA.e[i].y = pATA[indx+1];
		ATA.e[i].z = pATA[indx+2];

		U.e[i].x = pU[indx+0];
		U.e[i].y = pU[indx+1];
		U.e[i].z = pU[indx+2];
	}

	// �Ίp�s��̕��������Ƃ��āC�t�s��v�Z�̂��߂ɋt���ɂ��Ă���
	float l0 = (ATA.e[0].x <= 0.0) ? 0.0 : 1.0/sqrt(ATA.e[0].x);
	float l1 = (ATA.e[1].y <= 0.0) ? 0.0 : 1.0/sqrt(ATA.e[1].y);
	float l2 = (ATA.e[2].z <= 0.0) ? 0.0 : 1.0/sqrt(ATA.e[2].z);

	//// U^T M' U �̋t�s��v�Z
	matrix3x3 S1;
	S1.e[0].x = l0*U.e[0].x*U.e[0].x + l1*U.e[0].y*U.e[0].y + l2*U.e[0].z*U.e[0].z;
	S1.e[0].y = l0*U.e[0].x*U.e[1].x + l1*U.e[0].y*U.e[1].y + l2*U.e[0].z*U.e[1].z;
	S1.e[0].z = l0*U.e[0].x*U.e[2].x + l1*U.e[0].y*U.e[2].y + l2*U.e[0].z*U.e[2].z;
	S1.e[1].x = S1.e[0].x;
	S1.e[1].y = l0*U.e[1].x*U.e[1].x + l1*U.e[1].y*U.e[1].y + l2*U.e[1].z*U.e[1].z;
	S1.e[1].z = l0*U.e[1].x*U.e[2].x + l1*U.e[1].y*U.e[2].y + l2*U.e[1].y*U.e[2].z;
	S1.e[2].x = S1.e[0].z;
	S1.e[2].y = S1.e[1].z;
	S1.e[2].z = l0*U.e[2].x*U.e[2].x + l1*U.e[2].y*U.e[2].y + l2*U.e[2].z*U.e[2].z;

	R = Multiple(A, S1);	// R = A S^-1
	S = Multiple(Transpose(R), A); // S = R^-1 A = R^T A
}

//�z�񏉊���
__device__
	void MakeIdentity(matrix3x3 &M)
{
	M.e[0].x = 1.0;
	M.e[0].y = 0.0;
	M.e[0].z = 0.0;

	M.e[1].x = 0.0;
	M.e[1].y = 1.0;
	M.e[1].z = 0.0;

	M.e[2].x = 0.0;
	M.e[2].y = 0.0;
	M.e[2].z = 1.0;
}

//�]�u�s��
__device__
	matrix3x3 Transpose(const matrix3x3 &M)
{
	matrix3x3 T;

	T.e[0].x = M.e[0].x;
	T.e[0].y = M.e[1].x;
	T.e[0].z = M.e[2].x;

	T.e[1].x = M.e[0].y;
	T.e[1].y = M.e[1].y;
	T.e[1].z = M.e[2].y;

	T.e[2].x = M.e[0].z;
	T.e[2].y = M.e[1].z;
	T.e[2].z = M.e[2].z;

	return T;
}

//�t�s��
__device__
	matrix3x3 Inverse(const matrix3x3 &M)
{
	matrix3x3 I;

	I.e[0].x = M.e[0].x;
	I.e[0].y = M.e[1].x;
	I.e[0].z = M.e[2].x;

	I.e[1].x = M.e[0].y;
	I.e[1].y = M.e[1].y;
	I.e[1].z = M.e[2].y;

	I.e[2].x = M.e[0].z;
	I.e[2].y = M.e[1].z;
	I.e[2].z = M.e[2].z;

	float d = M.e[0].x*M.e[1].y*M.e[2].z- 
			 M.e[0].x*M.e[2].y*M.e[1].z+ 
			 M.e[1].x*M.e[2].y*M.e[0].z- 
			 M.e[1].x*M.e[0].y*M.e[2].z+ 
			 M.e[2].x*M.e[0].y*M.e[1].z- 
			 M.e[2].x*M.e[1].y*M.e[0].z;

	if(d == 0) d = 1;

	I.e[0].x =  (M.e[1].y*M.e[2].z-M.e[1].z*M.e[2].y)/d;
	I.e[0].y = -(M.e[0].y*M.e[2].z-M.e[0].z*M.e[2].y)/d;
	I.e[0].z =  (M.e[0].y*M.e[1].z-M.e[0].z*M.e[1].y)/d;
	I.e[1].x = -(M.e[1].x*M.e[2].z-M.e[1].z*M.e[2].x)/d;
	I.e[1].y =  (M.e[0].x*M.e[2].z-M.e[0].z*M.e[2].x)/d;
	I.e[1].z = -(M.e[0].x*M.e[1].z-M.e[0].z*M.e[1].x)/d;
	I.e[2].x =  (M.e[1].x*M.e[2].y-M.e[1].y*M.e[2].x)/d;
	I.e[2].y = -(M.e[0].x*M.e[2].y-M.e[0].y*M.e[2].x)/d;
	I.e[2].z =  (M.e[0].x*M.e[1].y-M.e[0].y*M.e[1].x)/d;

	return I;
}

__device__ 
	matrix3x3 Multiple(matrix3x3 &M1, matrix3x3 &M2)
{
	matrix3x3 M;

	M.e[0].x = M1.e[0].x * M2.e[0].x + M1.e[0].y * M2.e[1].x + M1.e[0].z * M2.e[2].x;
	M.e[0].y = M1.e[0].x * M2.e[0].y + M1.e[0].y * M2.e[1].y + M1.e[0].z * M2.e[2].y;
	M.e[0].z = M1.e[0].x * M2.e[0].z + M1.e[0].y * M2.e[1].z + M1.e[0].z * M2.e[2].z;
	 
	M.e[1].x = M1.e[1].x * M2.e[0].x + M1.e[1].y * M2.e[1].x + M1.e[1].z * M2.e[2].x;
	M.e[1].y = M1.e[1].x * M2.e[0].y + M1.e[1].y * M2.e[1].y + M1.e[1].z * M2.e[2].y;
	M.e[1].z = M1.e[1].x * M2.e[0].z + M1.e[1].y * M2.e[1].z + M1.e[1].z * M2.e[2].z;
	 
	M.e[2].x = M1.e[2].x * M2.e[0].x + M1.e[2].y * M2.e[1].x + M1.e[2].z * M2.e[2].x;
	M.e[2].y = M1.e[2].x * M2.e[0].y + M1.e[2].y * M2.e[1].y + M1.e[2].z * M2.e[2].y;
	M.e[2].z = M1.e[2].x * M2.e[0].z + M1.e[2].y * M2.e[1].z + M1.e[2].z * M2.e[2].z;

	return M;
}

__device__ 
	float3 Multiple(matrix3x3 &M1, float3 &V)
{
	float3 M;

	M.x = M1.e[0].x * V.x + M1.e[0].y * V.y + M1.e[0].z * V.z;
	M.y = M1.e[1].x * V.y + M1.e[1].y * V.y + M1.e[1].z * V.z;
	M.z = M1.e[2].x * V.z + M1.e[2].y * V.y + M1.e[2].z * V.z;

	return M;
}

#endif