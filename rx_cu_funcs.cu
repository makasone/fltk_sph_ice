#include "hip/hip_runtime.h"
/*! 
  @file rx_cu_funcs.cu
	
  @brief CUDA�֐� - �������֌W�Ȃ�

  @author Makoto Fujisawa
  @date 2009-08, 2011-06
*/
// FILE --rx_cu_funcs.cu--


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <cstdio>
#include <GL/glew.h>
#include <GL/glut.h>

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

#include <GL/freeglut.h>

#include "rx_cu_common.cu"



//-----------------------------------------------------------------------------
// CUDA�֐�
//-----------------------------------------------------------------------------
extern "C"
{
/*!
 * CUDA�f�o�C�X�̐ݒ�
 *  - �R�}���h���C�������Ɋ�Â�CUDA�f�o�C�X��ݒ�((��)-device 0)
 * @param[in] argc �R�}���h���C�������̐�
 * @param[in] argv �R�}���h���C���������X�g(argv[0]�͎��s�t�@�C����)
 */
void CuInit(int argc, char **argv)
{   
	if(checkCmdLineFlag(argc, (const char**)argv, "device")){
		int id = getCmdLineArgumentInt(argc, (const char**)argv, "device=");
		if(id < 0){
			id = gpuGetMaxGflopsDeviceId();
			hipSetDevice(id);
		}
		else{
			hipSetDevice(id);
		}
	}
	else{
		hipSetDevice( gpuGetMaxGflopsDeviceId() );
	}
}

/*!
 * CUDA�f�o�C�X�̐ݒ� - id�𒼐ڎw��
 * @param[in] id �f�o�C�XID
 */
void CuSetDevice(int id)
{ 
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	if(id < 0 || id >= device_count){
		id = gpuGetMaxGflopsDeviceId();
		hipSetDevice(0);
	}
	else{
		hipSetDevice(id);
	}
}

/*!
 * �f�o�C�X�������̊m��
 * @param[out] dPtr �f�o�C�X�������ւ̃|�C���^
 * @param[in] size �m�ۃT�C�Y(��������̃T�C�Y)
 */
void CuAllocateArray(void **dPtr, size_t size)
{
	RX_CUCHECK(hipMalloc(dPtr, size));
}

/*!
 * �f�o�C�X�������̉��
 * @param[in] devPtr �f�o�C�X�������ւ̃|�C���^
 */
void CuFreeArray(void *dPtr)
{
	RX_CUCHECK(hipFree(dPtr));
}

/*!
 * �f�o�C�X�������̈�̏�����
 * @param[in] dPtr �f�o�C�X�������ւ̃|�C���^
 * @param[in] val �����l
 * @param[in] size ����������̈�̃T�C�Y(��������̃T�C�Y)
 */
void CuSetArrayValue(void *dPtr, int val, size_t size)
{
	RX_CUCHECK(hipMemset(dPtr, val, size));
}

/*!
 * �f�o�C�X�������ԃR�s�[
 * @param[in] dDst �R�s�[��
 * @param[in] dSrc �R�s�[��
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayD2D(void *dDst, void *dSrc, int size)
{
	RX_CUCHECK(hipMemcpy(dDst, dSrc, size, hipMemcpyDeviceToDevice));
}


/*!
 * VBO���}�b�s���O
 * @param[in] vbo VBO,PBO��
 */
void *CuMapGLBufferObject(hipGraphicsResource **resource)
{
	void *ptr;
	RX_CUCHECK(hipGraphicsMapResources(1, resource, 0));
	size_t num_bytes;
	RX_CUCHECK(hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes, *resource));
	return ptr;
}

/*!
 * VBO���A���}�b�v
 * @param[in] vbo VBO,PBO��
 */
void CuUnmapGLBufferObject(hipGraphicsResource *resource)
{
	RX_CUCHECK(hipGraphicsUnmapResources(1, &resource, 0));
}

/*!
 * PBO,VBO�o�b�t�@��CUDA�ɓo�^
 * @param[in] vbo VBO,PBO��
 */
void CuRegisterGLBufferObject(uint vbo, hipGraphicsResource **resource)
{
	RX_CUCHECK(hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsNone));
}

/*!
 * PBO,VBO�o�b�t�@��CUDA����폜
 * @param[in] vbo VBO,PBO��
 */
void CuUnregisterGLBufferObject(hipGraphicsResource *resource)
{
	RX_CUCHECK(hipGraphicsUnregisterResource(resource));
}

/*!
 * �f�o�C�X����z�X�g�������ւ̃R�s�[
 * @param[in] hDst �R�s�[��z�X�g������(�Œ�size���m�ۂ���Ă��邱��)
 * @param[in] dSrc �R�s�[���f�o�C�X������
 * @param[in] vbo dSrc��VBO�̏ꍇ�CVBO��ID�D�����łȂ��ꍇ��0���w��
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayFromDevice(void* hDst, const void* dSrc, hipGraphicsResource **resource, int size)
{   
	if(resource) dSrc = CuMapGLBufferObject(resource);

	RX_CUCHECK(hipMemcpy(hDst, dSrc, size, hipMemcpyDeviceToHost));
	
	if(resource) CuUnmapGLBufferObject(*resource);
}

/*!
 * �z�X�g����f�o�C�X�������ւ̃R�s�[
 * @param[in] dDst �R�s�[��f�o�C�X������(�Œ�size���m�ۂ���Ă��邱��)
 * @param[in] hSrc �R�s�[���z�X�g������
 * @param[in] offset �R�s�[��I�t�Z�b�g
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayToDevice(void* dDst, const void* hSrc, int offset, int size)
{
	RX_CUCHECK(hipMemcpy((char*)dDst+offset, hSrc, size, hipMemcpyHostToDevice));
}

/*!
 * �X���b�h����
 */
void CuThreadSync(void)
{
	RX_CUCHECK(hipDeviceSynchronize());
}

/*!
 * �f�o�C�X�v���p�e�B�̕\��
 */
void CuDeviceProp(void)
{
	int n;	//�f�o�C�X��
	RX_CUCHECK(hipGetDeviceCount(&n));

	for(int i = 0; i < n; ++i){
		hipDeviceProp_t dev;

		// �f�o�C�X�v���p�e�B�擾
		RX_CUCHECK(hipGetDeviceProperties(&dev, i));

		printf("device %d\n", i);
		printf(" device name : %s\n", dev.name);
		printf(" total global memory : %d (MB)\n", dev.totalGlobalMem/1024/1024);
		printf(" shared memory / block : %d (KB)\n", dev.sharedMemPerBlock/1024);
		printf(" register / block : %d\n", dev.regsPerBlock);
		printf(" warp size : %d\n", dev.warpSize);
		printf(" max pitch : %d (B)\n", dev.memPitch);
		printf(" max threads / block : %d\n", dev.maxThreadsPerBlock);
		printf(" max size of each dim. of block : (%d, %d, %d)\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
		printf(" max size of each dim. of grid  : (%d, %d, %d)\n", dev.maxGridSize[0], dev.maxGridSize[1], dev.maxGridSize[2]);
		printf(" clock rate : %d (MHz)\n", dev.clockRate/1000);
		printf(" total constant memory : %d (KB)\n", dev.totalConstMem/1024);
		printf(" compute capability : %d.%d\n", dev.major, dev.minor);
		printf(" alignment requirement for texture : %d\n", dev.textureAlignment);
		printf(" device overlap : %s\n", (dev.deviceOverlap ? "ok" : "not"));
		printf(" num. of multiprocessors : %d\n", dev.multiProcessorCount);
		printf(" kernel execution timeout : %s\n", (dev.kernelExecTimeoutEnabled ? "on" : "off"));
		printf(" integrated : %s\n", (dev.integrated ? "on" : "off"));
		printf(" host memory mapping : %s\n", (dev.canMapHostMemory ? "on" : "off"));

		printf(" compute mode : ");
		if(dev.computeMode == hipComputeModeDefault) printf("default mode (multiple threads can use) \n");
		else if(dev.computeMode == hipComputeModeExclusive) printf("exclusive mode (only one thread will be able to use)\n");
		else if(dev.computeMode == hipComputeModeProhibited) printf("prohibited mode (no threads can use)\n");
		
	}

	printf("Device with Maximum GFLOPS : %d\n", gpuGetMaxGflopsDeviceId());
}

/*!
 * thrust::exclusive_scan�̌Ăяo��
 * @param[out] dScanData scan��̃f�[�^
 * @param[in] dData ���f�[�^
 * @param[in] num �f�[�^��
 */
void CuScan(unsigned int* dScanData, unsigned int* dData, unsigned int num)
{
	thrust::exclusive_scan(thrust::device_ptr<unsigned int>(dData), 
						   thrust::device_ptr<unsigned int>(dData+num),
						   thrust::device_ptr<unsigned int>(dScanData));
}

/*!
 * thrust::sort_by_key�ɂ��n�b�V���l�Ɋ�Â��\�[�g
 * @param[in] dHash �n�b�V���l
 * @param[in] dIndex �C���f�b�N�X(�p�[�e�B�N���C�|���S���Ȃ�)
 * @param[in] num �f�[�^��
 */
void CuSort(unsigned int *dHash, uint *dIndex, uint num)
{
	thrust::sort_by_key(thrust::device_ptr<unsigned int>(dHash),
						thrust::device_ptr<unsigned int>(dHash+num),
						thrust::device_ptr<unsigned int>(dIndex));
}




}   // extern "C"
