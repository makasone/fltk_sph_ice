#include "hip/hip_runtime.h"
//�e�N���X�^�̏�񂩂�ŏI�I�Ȍő̂̏����Z�o
//���͒P���ɕ���

#ifndef _GPU_ICE_STRUCTURE_H_
#define _GPU_ICE_STRUCTURE_H_

#include <iostream>
#include <hip/hip_runtime.h>
#include <rx_cu_common.cuh>

using namespace std;

#define EDGE 17

void LaunchCalcAverageGPU
	(
	int prtNum,
	float* sldPrtPos, 
	float* sldPrtVel, 
	float* sphPrtPos, 
	float* sphPrtVel, 
	float* smPrtPos, 
	float* smPrtVel,
	int* smIndxSet,
	int* PtoCIndx,
	int* PtoC,
	int PNumMax,
	int PtoCMax,
	int PtoCParamSize
	);

__global__ void CalcAverage(float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel, float* smPrtPos, float* smPrtVel, int* indxSet, int* PtoCIndx, int* PtoC, int PNumMax, int PtoCMax, int PtoCParamSize);

//__device__ int GetPtoCIndx(int pIndx);
//__device__ int GetPtoC(int pIndx, int lIndx, int oIndx);

void LaunchCalcAverageGPU(int prtNum, float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel, float* smPrtPos, float* smPrtVel, int* smIndxSet, int* PtoCIndx, int* PtoC, int PNumMax, int PtoCMax, int PtoCParamSize)
{
	int n = pow( prtNum, 1.0/3.0 ) + 0.5;	//�����̂̂P�ӂ̒��_��

	dim3 grid(n, n);
	dim3 block(n, 1, 1);

	//�^���v�Z
	CalcAverage<<<grid ,block>>>(sldPrtPos, sldPrtVel, sphPrtPos, sphPrtVel, smPrtPos, smPrtVel, smIndxSet, PtoCIndx, PtoC, PNumMax, PtoCMax, PtoCParamSize);
}

__global__
	void CalcAverage(float* sldPrtPos, float* sldPrtVel, float* sphPrtPos, float* sphPrtVel, float* smPrtPos, float* smPrtVel, int* indxSet, int* PtoCIndx, int* PtoC, int PNumMax, int PtoCMax, int PtoCParamSize)
{
	//�v�Z���闱�q�̔���
	int pIndx = blockIdx.x * EDGE * EDGE + blockIdx.y * EDGE + threadIdx.x;

	//���ꂼ��̃x�N�g�������������ς��Ƃ�
	float3 pos = make_float3(0.0f, 0.0f, 0.0f);
	float3 vel = make_float3(0.0f, 0.0f, 0.0f);
	float clusterNum = 0.0f;					//�N���X�^�̐�

	//TODO::�J�n�Y���C�I���Y��
	int pTocIndx = PtoCIndx[pIndx];

	for(int j = 0; j < pTocIndx; ++j)
	{
		//pIndx�Ԗڂ̗��q��������j�ڂ̃N���X�^
		int jcIndx = PtoC[(PtoCMax*PtoCParamSize) * pIndx + PtoCMax * 0 + j];
		int joIndx = PtoC[(PtoCMax*PtoCParamSize) * pIndx + PtoCMax * 1 + j];

		if(jcIndx == -1 || joIndx == -1){	continue;	}

		//�N���X�^jcIndx�Ԃ�joIndx�Ԗڂ̒��_
		int startIndx = indxSet[jcIndx*2+0];

		int smIndx = startIndx*3 + joIndx*3;

		pos.x += smPrtPos[smIndx+0];
		pos.y += smPrtPos[smIndx+1];
		pos.z += smPrtPos[smIndx+2];

		vel.x += smPrtVel[smIndx+0];
		vel.y += smPrtVel[smIndx+1];
		vel.z += smPrtVel[smIndx+2];

		clusterNum += 1.0f;
	}

	//�N���X�^�̐��Ŋ���
	if(clusterNum != 0.0f)
	{
		pos.x *= 1/clusterNum;
		pos.y *= 1/clusterNum;
		pos.z *= 1/clusterNum;

		vel.x *= 1/clusterNum;
		vel.y *= 1/clusterNum;
		vel.z *= 1/clusterNum;

		//pos.x /= clusterNum;
		//pos.y /= clusterNum;
		//pos.z /= clusterNum;

		//vel.x /= clusterNum;
		//vel.y /= clusterNum;
		//vel.z /= clusterNum;
	}		
	//�ǂ̃N���X�^�ɂ��܂܂�Ă��Ȃ��ꍇ�C�^����SPH�@�ɏ]��
	else
	{
		int sphIndx = pIndx*4;

		pos.x = sphPrtPos[sphIndx+0];
		pos.y = sphPrtPos[sphIndx+1];
		pos.z = sphPrtPos[sphIndx+2];

		vel.x = sphPrtVel[sphIndx+0];
		vel.y = sphPrtVel[sphIndx+1];
		vel.z = sphPrtVel[sphIndx+2];
	}

	//�ő̂̍ŏI�I�ȉ^���v�Z����
	int sldIndx = pIndx*3;

	sldPrtPos[sldIndx+0] = pos.x;
	sldPrtPos[sldIndx+1] = pos.y;
	sldPrtPos[sldIndx+2] = pos.z;

	sldPrtVel[sldIndx+0] = vel.x;
	sldPrtVel[sldIndx+1] = vel.y;
	sldPrtVel[sldIndx+2] = vel.z;

	//�K���ɐ��`��Ԃ��ǂ�
	sphPrtPos[pIndx*4+0] = pos.x;
	sphPrtPos[pIndx*4+1] = pos.y;
	sphPrtPos[pIndx*4+2] = pos.z;

	sphPrtVel[pIndx*4+0] = vel.x;
	sphPrtVel[pIndx*4+1] = vel.y;
	sphPrtVel[pIndx*4+2] = vel.z;
}

//__device__ int GetPtoCIndx(int pIndx)
//{
//	return 0;
//}
//
//__device__ int GetPtoC(int pIndx, int lIndx, int oIndx)
//{
//	return 0;
//}


#endif